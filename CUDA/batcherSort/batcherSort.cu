#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cmath>
#define THREADS_PER_BLOCK 32 
#define SIZE 8 

__global__ void bitonicSorterHalver(int *list, int j) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        int p_idx = idx ^ j;

        if (idx < p_idx) {
                if (list[idx] > list[p_idx]) {
                    int temp = list[idx];
                    list[idx] = list[p_idx];
                    list[p_idx] = temp;
                }
        }
}

int main(void){
        int n = SIZE;

        int bitonic_list[n] = {1,2,3,6,7,4,2,1};
        for( int i = 0 ; i < n ; ++i )
                std::cout << bitonic_list[i] << " ";

        int * device_bitonic_list; 
        hipMalloc((void**)&device_bitonic_list,sizeof(int)*n);
        hipMemcpy(device_bitonic_list,bitonic_list,sizeof(int)*n,hipMemcpyHostToDevice);

        int threads = std::min(n,THREADS_PER_BLOCK);
        int blocks = (n + threads - 1) / threads;

        for (int j = n / 2 ; j > 0 ; j /= 2) {
            bitonicSorterHalver<<<blocks, threads>>>(device_bitonic_list, j);
            hipDeviceSynchronize();
        }

        hipMemcpy(bitonic_list,device_bitonic_list,sizeof(int)*n,hipMemcpyDeviceToHost);
        hipFree(device_bitonic_list);

        printf("\nAfter Bitonic Sorting: \n");
        for( int i = 0 ; i < n ; ++i )
                std::cout << bitonic_list[i] << " ";
        std::cout << std::endl;

        return 0;
}
